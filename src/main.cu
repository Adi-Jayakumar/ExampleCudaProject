
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void AddIntsCUDA(int* a, int* b)
{
    a[0] += b[0];
}

int main()
{
    
    int a = 5;
    int b = 9;
    int* d_a, *d_b;
    if(hipMalloc(&d_a, sizeof(int)) != hipSuccess)
        std::cout << "OH NO" << std::endl;
    if(hipMalloc(&d_b, sizeof(int)) != hipSuccess)
        std::cout << "OH NO" << std::endl;
    
    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
    AddIntsCUDA<<<1,1>>>(d_a, d_b);
    
    hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);
    std::cout << "ANS: " << a << std::endl;
    hipFree(d_a);
    hipFree(d_b);
    return 0;
}